#include "hip/hip_runtime.h"
// by Stefan Vocht
// this file contains the CUDA implementation of functions related to the matrix exponential used to compute 
// the scores and gradients for training a MHN on data where the sample ages are known


#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hipblas.h>

#include <cmath>

#include "cuda_state_space_restriction.cuh"


double calc_gamma(hipblasHandle_t handle, const double *theta, int n, const State *state, double *dg){
    int mutation_num = get_mutation_num(state);
    int nx = 1 << mutation_num;
    int block_num, thread_num;
    double gamma;
    determine_block_thread_num(block_num, thread_num);

    hipMemset(dg, 0, nx * sizeof(double));
    cuda_subtract_q_diag(theta, state, n, mutation_num, dg, block_num, thread_num);
    hipblasDnrm2(handle, nx, dg, 1, &gamma);
    return gamma;
}


// TODO optimize this later
__global__ void zero_mask(double *arr, int k, int size){
    int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = cuda_index; j < size; j += stride) {
		arr[k] = ((j >> k) & 1) * arr[k];
	}
}


void dua(hipblasHandle_t handle, const double *theta, int n, double *bq, const State *state, double t, int i, int k, double eps, double gamma, double dgamma, double *pt, double *dp, double *dq, double *tmp, double *tmp2, int count_before_i){

    int mutation_num = get_mutation_num(state);
    int nx = 1 << mutation_num;
    int nn = 0;

    int block_num, thread_num;
    determine_block_thread_num(block_num, thread_num, mutation_num);

    hipMemset(pt, 0, nx * sizeof(double));
    hipMemset(dp, 0, nx * sizeof(double));
    hipMemset(dq, 0, nx * sizeof(double));

    double gfac = 1.;
    double dgam_inv = -1./(gamma*gamma) * dgamma;
    double gam_inv = 1 / gamma;
    double ewg = exp(-1. * gamma * t);
    double mass_defect = 0.0;

    while (eps < (1 - mass_defect)){
        mass_defect += ewg;

        hipblasDaxpy(handle, nx, &ewg, bq, 1, pt, 1);

        hipblasDaxpy(handle, nx, &ewg, dq, 1, dp, 1);

        gfac = ewg*dgamma*(nn/gamma - t);
        hipblasDaxpy(handle, nx, &gfac, bq, 1, dp, 1);

        nn += 1;

        cuda_q_vec(theta, bq, state, tmp, n, mutation_num, true, false);
        hipblasDscal(handle, nx, &dgam_inv, tmp, 1);
        hipMemset(tmp2, 0, nx * sizeof(double));
        cuda_restricted_kronvec<<<block_num, thread_num, n * sizeof(double)>>>(theta, i, bq, state, true, false, n, mutation_num, count_before_i, tmp2);
        zero_mask<<<block_num, thread_num>>>(tmp2, k, nx);
        hipblasDaxpy(handle, &gam_inv, tmp2, 1, tmp, 1);

        cuda_q_vec(theta, dq, state, tmp2, n, mutation_num, true, false);
        hipblasDaxpy(handle, nx, &gam_inv, tmp2, 1, dq, 1);
        add_arrays<<<block_num, thread_num>>>(tmp, dq, nx);

        cuda_q_vec(theta, bq, state, tmp, n, mutation_num, true, false);
        hipblasDaxpy(handle, nx, &gam_inv, tmp, 1, bq, 1);

        ewg *= gamma*t / nn;
    }
}


int empirical_distribution_index(const State *current_state, const State *former_state){
    int current_mutation_num = get_mutation_num(current_state);
    int xk_index = (1 << current_mutation_num) - 1;
    int bit_setter = 1;

    uint32_t state_copy_current = current_state->parts[0];
    uint32_t state_copy_former = former_state->parts[0];

    for(int j = 0; j < 32 * STATE_SIZE; j++){
        if (state_copy_current & 1){
            if (!(state_copy_former & 1)){
                xk_index &= ~bit_setter;
            }
            bit_setter <<= 1;
        }
        if ((j+1) & 31){
            state_copy_current >>= 1;
            state_copy_former >>= 1;
        } else {
            state_copy_current = current_state->parts[(j+1) >> 5];
            state_copy_former = former_state->parts[(j+1) >> 5];
        }
    }
    return xk_index;
}


extern "C" 
{
    int DLL_PREFIX cuda_gradient_and_score_dua(const double *ptheta, int n, const State *mutation_data, const double *ages, int data_size, double eps, double *grad_out, double *score_out){
        
        int max_mutation_num = 0;
        for (int i = 0; i < data_size; i++) {
            if (get_mutation_num(&mutation_data[i]) > max_mutation_num) max_mutation_num = get_mutation_num(&mutation_data[i]);
        }

        int nx = 1 << max_mutation_num;

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        double *bq, *dq;
        double *pt, *dp;
        double *tmp, *tmp2;
        double *cuda_grad, *cuda_score;
        double *cuda_theta;
        double *dg, *deriv_dg;
        double *cuda_dgamma;
        double dgamma;

        int block_num, thread_num;

        hipMalloc(&bq, nx * sizeof(double));
        hipMalloc(&pt, nx * sizeof(double));
        hipMalloc(&dp, nx * sizeof(double));
        hipMalloc(&dq, nx * sizeof(double));
        hipMalloc(&dg, nx * sizeof(double));
        hipMalloc(&deriv_dg, nx * sizeof(double));
        hipMalloc(&tmp, nx * sizeof(double));
        hipMalloc(&tmp2, nx * sizeof(double));
        hipMalloc(&cuda_grad, n*n * sizeof(double));
        hipMalloc(&cuda_score, sizeof(double));
        hipMalloc(&cuda_theta, n*n * sizeof(double));
        hipMalloc(&cuda_dgamma, sizeof(double));

        hipMemcpy(cuda_theta, ptheta, n*n * sizeof(double), hipMemcpyHostToDevice);


        for (int k = 1; k < data_size; k++){
            int current_mutation_num = get_mutation_num(&mutation_data[k]);
            int current_nx = 1 << current_mutation_num;
            int current_nx_half = current_nx / 2;

            determine_block_thread_num(block_num, thread_num, current_mutation_num);

            hipMemset(bq, 0, current_nx * sizeof(double));

            int xk_index = empirical_distribution_index(&mutation_data[k], &mutation_data[k-1]);
            fill_array<<<1, 1>>>(bq + xk_index, 1.0, sizeof(double));

            double t = ages[k] - ages[k-1];

            double gamma = calc_gamma(handle, cuda_theta, n, &mutation_data[k], dg);

            uint32_t state_copy_i = mutation_data[k]->parts[0];
            int count_before_i = 0;  // counts the number of mutations that occured before the ith index

            for (int i = 0; i < n; i++){

                // compute the derivative of the diagonal using the shuffle trick
                hipMemset(deriv_dg, 0, current_nx * sizeof(double));
                cuda_subdiag<<<block_num, thread_num>>>(theta, &mutation_data[k], i, n, current_mutation_num, deriv_dg);
                multiply_arrays_elementwise<<<block_num, thread_num>>>(dg, deriv_dg, current_nx);

                uint32_t state_copy_j = mutation_data[k]->parts[0];

                for(int j = 0; j < n; j++){
                    if (state_copy_j & 1){
                        // shuffle deriv_dg
                        hipblasDcopy(handle, current_nx_half, deriv_dg, 2, dq, 1);
                        hipblasDcopy(handle, current_nx_half, deriv_dg+1, 2, dq + current_nx_half, 1);
                        hipblasDcopy(handle, current_nx, dq, 1, deriv_dg, 1);   // this should be optimized by switching pointers

                        if (i == j){
                            sum_over_array<<<block_num, thread_num, thread_num * sizeof(double)>>>(deriv_dg, dq, current_nx);
                            sum_over_array<<<1, block_num, block_num * sizeof(double)>>>(dq, cuda_dgamma, block_num);
                            hipMemcpy(&dgamma, cuda_dgamma, sizeof(double), hipMemcpyDeviceToHost);
                        } else {
                            sum_over_array<<<block_num, thread_num, thread_num * sizeof(double)>>>(deriv_dg + current_nx_half, dq, current_nx_half);
                            sum_over_array<<<1, block_num, block_num * sizeof(double)>>>(dq, cuda_dgamma, block_num);
                            hipMemcpy(&dgamma, cuda_dgamma, sizeof(double), hipMemcpyDeviceToHost);
                        }

                    } else if (i == j){
                        sum_over_array<<<block_num, thread_num, thread_num * sizeof(double)>>>(deriv_dg, dq, current_nx);
                        sum_over_array<<<1, block_num, block_num * sizeof(double)>>>(dq, cuda_dgamma, block_num);
                        hipMemcpy(&dgamma, cuda_dgamma, sizeof(double), hipMemcpyDeviceToHost);
                    }

                    if ((state_copy_j & 1) || i == j){
                        dua(handle, cuda_theta, n, bq, &mutation_data[k], t, i, j, eps, gamma, dgamma, pt, dp, dq, tmp, tmp2, count_before_i);
                        // add result to gradient
                        divide_arrays_elementwise<<<1, 1>>>(dp + current_nx - 1, pt + current_nx - 1, dp + current_nx - 1, 1);
                        add_arrays<<<1, 1>>>(dp + current_nx - 1, cuda_grad + i*n + j, 1);
                    }

                    // if the mutation state of the next gene is stored on the current state_copy_j, make a bit shift to the right
                    // else state_copy_j becomes the next integer stored in the given state (x >> 5  <=> x // 32, x & 31 <=> x % 32)
                    if ((j + 1) & 31){
                        state_copy_j >>= 1;
                    }
                    else {
                        state_copy_j = state->parts[(j + 1) >> 5];
                    }
                }

                count_before_i += (state_copy_i & 1);

                if ((i + 1) & 31){
                     state_copy_i >>= 1;
                }
                else {
                    state_copy_i = state->parts[(i + 1) >> 5];
                }
            }
            // update total score
            add_to_score<<<1, 1>>>(cuda_score, pt + current_nx - 1);
        }

        hipMemcpy(grad_out, cuda_grad, n*n * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(score_out, cuda_score, sizeof(double), hipMemcpyDeviceToHost);

        hipFree(bq);
        hipFree(dq);
        hipFree(dg);
        hipFree(deriv_dg);
        hipFree(pt);
        hipFree(dp);
        hipFree(tmp);
        hipFree(tmp2);
        hipFree(cuda_grad);
        hipFree(cuda_score);
        hipFree(cuda_theta);
        hipFree(cuda_dgamma);

        hipblasDestroy(handle);
    }

}