#include "hip/hip_runtime.h"
﻿
// by Stefan Vocht
// this file contains the CUDA implementation of State Space Restrictions used to compute 
// the gradients for training a MHN


#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <math.h>
#include <chrono>
#include <iostream>


// on Windows we need to add a prefix in front of the function we want to use in other code
// on Linux this is not needed, so we define DLL_PREFIX depending on which os this code is compiled on
#ifdef _WIN32
#define DLL_PREFIX __declspec(dllexport)
#else
#define DLL_PREFIX 
#endif


// this struct is used to store states representing up to 32 * STATE_SIZE genes
// STATE_SIZE must be defined during compilation
typedef struct {
     unsigned int parts[STATE_SIZE];
} State;


/**
 * Counts number of 1s in binary representation of number x, where x is a 32-bit integer
 * Source: https://stackoverflow.com/questions/109023/how-to-count-the-number-of-set-bits-in-a-32-bit-integer
 *
 * @param[in] i the integer of which we want to count the number of set bits
*/
int count_ones32(uint32_t i){
	i = i - ((i >> 1) & 0x55555555);        			// add pairs of bits
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);  	// quads
    i = (i + (i >> 4)) & 0x0F0F0F0F;        			// groups of 8
    return (i * 0x01010101) >> 24;          			// horizontal sum of bytes
}


/**
 * Counts number of 1s in binary representation of number x, where x is a 64-bit integer
 * Source: https://en.wikipedia.org/wiki/Hamming_weight 
*
 * @param[in] i the int64 of which we want to count the number of set bits
*/
int count_ones(uint64_t x) {
    x -= (x >> 1) & 0x5555555555555555LL;             					//put count of each 2 bits into those 2 bits
    x = (x & 0x3333333333333333LL) + ((x >> 2) & 0x3333333333333333LL); //put count of each 4 bits into those 4 bits 
    x = (x + (x >> 4)) & 0x0f0f0f0f0f0f0f0fLL;        					//put count of each 8 bits into those 8 bits 
    return (x * 0x0101010101010101LL) >> 56;  							//returns left 8 bits of x + (x<<8) + (x<<16) + (x<<24) + ... 
}


// computes the number of mutations present in a given state
/**
 * Computes the number of mutations present in a given state
 *
 * @param[in] state A pointer to a State of which we want to count the number of mutations it contains
*/
int get_mutation_num(const State *state){
	int mutation_num = 0;
	for(int i = 0; i < STATE_SIZE; i++){
		mutation_num += count_ones32(state->parts[i]);
	}
	return mutation_num;
}


/**
 * we determine the number of blocks and threads used in the CUDA kernels for the current data point with this function
 *
 * @param[out] block_num number of blocks that should be used for the CUDA kernels
 * @param[out] thread_num number of threads that should be used for the CUDA kernels
 * @param[in] mutation_num number of mutations present in the current state
*/
inline void determine_block_thread_num(int &block_num, int &thread_num, const int mutation_num) {

	// block_num and thread_num have to be powers of two, else cuda_restricted_kronvec will not work
	// maximum 256 blocks with 1024 threads
	if (mutation_num >= 17) {
		block_num = 256;
		thread_num = 512;
	}
	// minimum 32 * STATE_SIZE threads, else for n = 32 * STATE_SIZE (which is the maximum possible n) not all thetas get loaded in kron_vec
	else if (mutation_num < 12) {
		block_num = 32;
		thread_num = 64;
	}
	else {
		block_num = 1 << (mutation_num / 2);
		thread_num = 1 << (mutation_num / 2 + (mutation_num & 1));
	}
}

/**
 * this function is the cuda implementation of the kronvec function for state space restriction
 *
 * IMPORTANT: the result is added to the entries of pout! This makes the q_vec function more efficient. 
 * If you need the result without adding, initialize pout with zeros.
 *
 * @param[in] ptheta array containing the values of theta
 * @param[in] i vector is multiplied with the ith kronecker product (ith summand in eq. 9 of the original paper) 
 * @param[in] px vector that is multiplied with the kronecker product
 * @param[in] state current state used to compute the gradient
 * @param[in] diag if false, the diagonal of the kronecker product is set to zero
 * @param[in] transp if true, the kronecker product is transposed
 * @param[in] n total number of genes considered by the MHN, also column and row size of theta
 * @param[in] mutation_num number of mutations present in state
 * @param[in] count_before_i number of genes mutated that have a lower index than i
 * @param[out] pout vector which will contain the result of this multiplication
*/
__global__ void cuda_restricted_kronvec(const double* __restrict__ ptheta, const int i, const double* __restrict__ px, const State state, const bool diag, const bool transp, const int n, const int mutation_num, const int count_before_i, double* __restrict__ pout) {
	const int stride = blockDim.x * gridDim.x;
	const int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	// in the following 1 << i is equivalent to 2^i, x >> i is equivalent to x // 2^i, x & ((1<<i)-1) to x % 2^i
	const int nx = 1 << mutation_num;
	const int nxhalf = nx >> 1;

	extern __shared__ double theta_i[];

	// tells us, if state i is set to 1
	int state_i_one = (state.parts[i >> 5] >> (i & 31)) & 1;

	if (!diag && !state_i_one) {
		// in this case the result is zero in every entry
		// this means we do not have to add anything to pout and can just return
		return;
	}

	// load the ith row of theta into shared memory for more efficient access
	for(int j = threadIdx.x; j < n; j += blockDim.x){
		theta_i[j] = ptheta[i*n + j];
	}

	__syncthreads();


	// patch_size is important for later for the case i == j in the shuffle algorithm
	// as we do not actually shuffle the data in px in this implementation (only implicitly), we have to keep track of some indices
	// and which entries have to be computed together in the case i == j. Those two entries are px1 and px2
	// the index difference between those is patch_size (patches, as over all, the px1 and px2 occur in patches of size 2^z
	// z here is the number of events/bits set to 1 that have an index smaller than i)
	const int patch_size = 1 << count_before_i;
	int x_index1 = (cuda_index >> count_before_i) * 2 * patch_size + (cuda_index & (patch_size - 1));
	int x_index2 = x_index1 + patch_size;

	while(x_index2 < nx){
		double px1 = px[x_index1];
		double px2 = px[x_index2];

		int state_copy = state.parts[0];

		for (int j = 0; j < n; j++) {
			if (state_copy & 1) {
				// change the indices as they would change if we did an actual shuffle
				x_index1 = (x_index1 >> 1) + (x_index1 & 1) * nxhalf;
				x_index2 = (x_index2 >> 1) + (x_index2 & 1) * nxhalf;
				double theta = theta_i[j];

				if (i == j) {
					// at the beginning, x_index_1 and x_index_2 were chosen in such a way that for i == j those are the entries that are added/multiplied together
					// we want to know which entry is in which "column" (see original shuffle algorithm), they are loaded accordingly in right_side/left_side
					// here we prevent "if" diverging branches, which would lead to serial execution, by implicitly doing the if clause while computing right_side
					// as left_side must be the px that is not right_side, we get it with px1 + px2 - right_side (again no branching)
					double right_side, left_side;
					right_side = px1 + (x_index2 > x_index1) * (px2 - px1);
					left_side = px1 + px2 - right_side;

					// this part is pratically the same as in the original kronvec function
					if (!transp) {
						right_side = left_side * theta;
						if (diag) {
							left_side = -right_side;
						}
						else {
							left_side = 0;
						}
					}
					else {
						if (diag) {
							left_side = (right_side - left_side) * theta;
						}
						else {
							left_side = right_side * theta;
						}
						right_side = 0;
					}

					// update values of px1,px2 with the values from right_side/left_side (again with implicit if condition)
					px1 = right_side + (x_index1 < x_index2) * (left_side - right_side);
					px2 = right_side + left_side - px1;
				}
				else {
					// this is equivalent to "if(x_index >= nxhalf) px *= theta_i[j] else px *= 1
					// we dont use a if clause to prevent diverging branches
					px1 *= 1 + (x_index1 >= nxhalf) * (theta - 1);
					px2 *= 1 + (x_index2 >= nxhalf) * (theta - 1);
				}
			} 
			else if (i == j) {
				// if the ith gene is not mutated, we simply multiply the entries with (-theta_ii)
				px1 *= -theta_i[i];
				px2 *= -theta_i[i];
			}

			// if the mutation state of the next gene is stored on the current state_copy, make a bit shift to the right
			// else state_copy becomes the next integer stored in the given state (x >> 5  <=> x // 32, x & 31 <=> x % 32)
			if ((j + 1) & 31){
				state_copy >>= 1;
			}
			else {
				state_copy = state.parts[(j + 1) >> 5];
			}
			
		}
		// add the px values to the output array
		pout[x_index1] += px1;
		pout[x_index2] += px2;

		// if patch_size is bigger than stride, we have to do corrections to the indices
		if(stride < patch_size){
			// check if the current index is inside an odd patch, if so, jump to the next one
			x_index1 += stride;
			x_index1 += ((x_index1 >> count_before_i) & 1) * patch_size;
			x_index2 = x_index1 + patch_size;
		} else {
			x_index1 += 2*stride;
			x_index2 += 2*stride;
		}
	}
}


/**
 * computes y = Q(ptheta) * x, result is saved in yout
 *
 * important: ptheta, x and yout must be allocated using hipMalloc()!
 *
 * @param[in] ptheta array containing the theta entries
 * @param[in] x vector that should be multiplied with Q(ptheta)
 * @param[in] state state representing current tumor sample
 * @param[out] yout array in which the result is stored
 * @param[in] n number of genes considered by the MHN, also number of columns/rows of theta
 * @param[in] mutation_num number of mutations present in the current state / tumor sample
 * @param[in] diag if false, the diag of Q is set to zero during multiplication
 * @param[in] transp if true, multiplication is done with the transposed Q
*/
static void cuda_q_vec(const double *ptheta, const double *x, const State *state, double *yout, const int n, const int mutation_num, const bool diag, const bool transp) {
	
	const int nx = 1 << mutation_num;
	hipMemset(yout, 0, nx * sizeof(double));

	int block_num, thread_num;
	int mutation_counter = 0;

	determine_block_thread_num(block_num, thread_num, mutation_num);

	for (int i = 0; i < n; i++) {
		// this would also be done in the kernel, but its faster to check it here
		if (!((state->parts[i >> 5] >> (i & 31)) & 1) && !diag) continue;

		cuda_restricted_kronvec<<<block_num, thread_num, n * sizeof(double)>>>(ptheta, i, x, *state, diag, transp, n, mutation_num, mutation_counter, yout);
		mutation_counter++;
	}
}


/**
 * computes the ith subdiagonal of Q and subtracts(!) it from dg
 * we subtract it, because in jacobi() we need 1 - dg, so dg is initialized with 1 and we subtract the subdiags
 *
 * @param[in] ptheta array containing the theta entries
 * @param[in] state state representing current tumor sample
 * @param[in] i this function computes the ith subdiagonal
 * @param[in] n number of genes considered by the MHN, also number of columns/rows of theta
 * @param[in] mutation_num number of mutations present in the current state / tumor sample
 * @param[in, out] dg the subdiagonal is subtracted from the values in this array
*/
__global__ void cuda_subdiag(const double *ptheta, const State state, const int i, const int n, const int mutation_num, double *dg) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	// in the following 1 << i is equivalent to 2^i, x >> i is equivalent to x // 2^i, x & ((1<<i)-1) to x % 2^i
	const int nx = 1 << mutation_num;

	// store the ith row of theta in shared memory for more efficient access
	extern __shared__ double theta_i[];

	for(int j = threadIdx.x; j < n; j += blockDim.x){
		theta_i[j] = ptheta[i*n + j];
	}
	__syncthreads();

	
	for (int k = cuda_index; k < nx; k += stride) {

		double dg_entry = 1;

		int state_copy = state.parts[0];
		int position_condition = k;
		for (int j = 0; j < n; j++) {
			double theta = theta_i[j];
			// depending on the index different thetas have to be multiplied to the subdiag entry
			if (state_copy & 1) {
				if (i == j) {
					dg_entry *= -(1 - (position_condition & 1)) * theta;
				}
				else {
					dg_entry *= 1 + (position_condition & 1) * (theta - 1);
				}

				position_condition >>= 1;
			}
			else if (i == j) {
				dg_entry *= -theta;
			}

			// if the mutation state of the next gene is stored on the current state_copy, make a bit shift to the right
			// else state_copy becomes the next integer stored in the given state (x >> 5  <=> x // 32, x & 31 <=> x % 32)
			if ((j + 1) & 31){
				state_copy >>= 1;
			}
			else {
				state_copy = state.parts[(j + 1) >> 5];
			}
		}
		//subtract the subdiagonal from the diagonal entries
		dg[k] -= dg_entry;
	}
}



/**
 * subtracts the diag of q from the given dg array, result can be found in dg
 *
 * @param[in] ptheta array containing the theta entries
 * @param[in] state state representing current tumor sample
 * @param[in] n number of genes considered by the MHN, also number of columns/rows of theta
 * @param[in] mutation_num number of mutations present in the current state / tumor sample
 * @param[in, out] dg the subdiagonals are subtracted from the values in this array
 * @param[in] block_num number of blocks used for the CUDA kernels
 * @param[in] thread_num  number of threads used for the CUDA kernels
*/
static void cuda_subtract_q_diag(const double *ptheta, const State *state, const int n, const int mutation_num, double *dg, int block_num, int thread_num) {
	for (int i = 0; i < n; i++) {
		cuda_subdiag<<<block_num, thread_num, n * sizeof(double)>>>(ptheta, *state, i, n, mutation_num, dg);
	}
}


__global__ void fill_array(double *arr, double x, const int size){
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = cuda_index; k < size; k += stride) {
		arr[k] = x;
	}
}

__global__ void add_arrays(const double *arr1, double *arr_inout, const int size) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = cuda_index; k < size; k += stride) {
		arr_inout[k] += arr1[k];
	}
}

__global__ void divide_arrays_elementwise(const double *arr1, const double *arr2, double *out, const int size) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = cuda_index; k < size; k += stride) {
		out[k] = arr1[k] / arr2[k];
	}
}

__global__ void multiply_arrays_elementwise(const double *arr1, double *arr_inout, const int size) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = cuda_index; k < size; k += stride) {
		arr_inout[k] *= arr1[k];
	}
}

/**
 * this functions multiplies [I-Q]^(-1) with b
 * all arrays given to this function must be allocated using hipMalloc()
 *
 * @param[in] ptheta array containing the theta entries
 * @param[in] b array that is multiplied with [I-Q]^(-1)
 * @param[in] state state representing current tumor sample
 * @param[in] mutation_num number of mutations present in the current state / tumor sample
 * @param[in] transp if true, b is multiplied with the tranposed [I-Q]^(-1)
 * @param[in] n number of genes considered by the MHN, also number of columns/rows of theta
 * @param[out] xout the results of this functio are stored in this array
 * @param[in, out] tmp this array is used to store temporary data, has to have size 2^mutation_num
 * @param[in, out] dg this array stores the diagonal of [I-Q], must have size 2^mutation_num
*/
static void cuda_jacobi(const double *ptheta, const double *b, const State *state, const int mutation_num, const bool transp, const int n, double *xout, double *tmp, double *dg) {

	const int nx = 1 << mutation_num;

	int block_num, thread_num;
	determine_block_thread_num(block_num, thread_num, mutation_num);

	// initialize the diagonal entries and xout
	fill_array<<<block_num, thread_num >>>(dg, 1, nx);
	cuda_subtract_q_diag(ptheta, state, n, mutation_num, dg, block_num, thread_num);
	fill_array<<<block_num, thread_num >>>(xout, 1. / (1. * nx), nx);

	// compute the product of [I-Q]^(-1) with b
	for (int z = 0; z < mutation_num + 1; z++) {
		cuda_q_vec(ptheta, xout, state, tmp, n, mutation_num, false, transp);
		add_arrays<<<block_num, thread_num >>>(b, tmp, nx);
		divide_arrays_elementwise<<<block_num, thread_num >>>(tmp, dg, xout, nx);
	}
}

/**
 * this functions shuffles the entries of old_vec into the entries of to_shuffle_vec
 *
 * @param[in] old_vec array that should be shuffled
 * @param[out] to_shuffle_vec array in which the shuffled vector is stored
 * @param[in] nx size of both vectors
*/
__global__ void shuffle(const double* __restrict__ old_vec, double* __restrict__ to_shuffle_vec, const int nx) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = cuda_index; k < nx; k += stride) {
		int greater_than_nx = (k >= nx / 2);
		to_shuffle_vec[k] = old_vec[2 * (k - greater_than_nx * nx / 2) + greater_than_nx];
	}
}


/**
 * inspired by https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
 * computes the sum of all entries in a given array
*/
__global__ void sum_over_array(const double *arr, double *result, int size) {

	extern __shared__ double sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	double partial_sum = 0;

	for (unsigned int s = i; s < size; s += stride) {
		partial_sum += arr[s];
	}

	sdata[tid] = partial_sum;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) result[blockIdx.x] = sdata[0];
}


__global__ void print_vec(double *vec, int size) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = cuda_index; k < size; k++) {
		printf("%g, ", vec[k]);
	}
	printf("\n\n");
}


/**
 * compute the gradient for one tumor sample
 *
 * @param[in] ptheta array containing the theta entries
 * @param[in] state state representing current tumor sample
 * @param[in] n number of genes considered by the MHN, also number of columns/rows of theta
 * @param[out] grad array that will contain the gradient at the end, size n*n
 * @param[in] p0_pD memory buffer needed for this function, size 2^mutation_num
 * @param[in] pth memory buffer needed for this function, size 2^mutation_num
 * @param[in] q memory buffer needed for this function, size 2^mutation_num
 * @param[in] tmp1 memory buffer needed for this function, size 2^mutation_num
 * @param[in] tmp2 memory buffer needed for this function, size 2^mutation_num
*/
static void cuda_restricted_gradient(const double *ptheta, const State *state, const int n, double *grad, double *p0_pD, double *pth, double *q, double *tmp1, double *tmp2) {

	// get the number of mutated genes in the current sample and compute the size of the memory buffers
	const int mutation_num = get_mutation_num(state);
	const int nx = 1 << mutation_num;

	const double one = 1;

	// set all entries of p0_pD to zero, set the first entry to one
	hipMemset(p0_pD, 0, nx * sizeof(double));
	hipMemcpy(p0_pD, &one, sizeof(double), hipMemcpyHostToDevice);

	cuda_jacobi(ptheta, p0_pD, state, mutation_num, false, n, pth, tmp1, tmp2);

	// set all entries of p0_pD to zero, set the last entry to 1/pth[last_index]
	hipMemset(p0_pD, 0, sizeof(double));
	hipMemcpy(p0_pD + nx - 1, &one, sizeof(double), hipMemcpyHostToDevice);
	divide_arrays_elementwise<<<1, 1>>>(p0_pD + nx - 1, pth + nx - 1, p0_pD + nx - 1, 1);

	cuda_jacobi(ptheta, p0_pD, state, mutation_num, true, n, q, tmp1, tmp2);

	double *old_vec, *shuffled_vec, *swap_vec;
	int block_num, thread_num;

	determine_block_thread_num(block_num, thread_num, mutation_num);

	// initialize grad with zeros
	hipMemset(grad, 0, n*n * sizeof(double));

	// this counter is used for cuda_restricted_kronvec and counts how many of the genes
	// up to this point have been mutated in the tumor sample
	int kronvec_count_before_i = 0;

	for (int i = 0; i < n; i++) {
		hipMemset(tmp1, 0, nx * sizeof(double));

		// check if the current gene is mutated
		if((state->parts[i >> 5] >> (i & 31)) & 1){
			cuda_restricted_kronvec<<<block_num, thread_num, n*sizeof(double)>>>(ptheta, i, pth, *state, true, false, n, mutation_num, kronvec_count_before_i, tmp1);
			kronvec_count_before_i++;
		}
		else{
			// if the current gene is not mutated we can set the parameter count_before_i to mutation_num-1 to get a patch_size as large as possible
			// this gives us better aligned memory access on global memory
			cuda_restricted_kronvec<<<block_num, thread_num, n*sizeof(double)>>>(ptheta, i, pth, *state, true, false, n, mutation_num, mutation_num - 1, tmp1);
		}

		// tmp1 contains the result of the call to cuda_restricted_kronvec above
		multiply_arrays_elementwise<<<block_num, thread_num>>>(q, tmp1, nx);

		old_vec = tmp1;
		shuffled_vec = tmp2;
		int state_copy = state->parts[0];
		double *grad_i = grad + i * n;

		// use the shuffle trick for a more efficient computation of the gradient
		for (int j = 0; j < n; j++) {
			// confusion warning: the p0_pD here has nothing to do with p0 or pD
			// in this section p0_pD is used again, because we need an allocated array and p0_pD isnt needed anymore so we can just use that as memory
			if (state_copy & 1) {
				shuffle<<<block_num, thread_num>>>(old_vec, shuffled_vec, nx);
				if (i == j) {
					sum_over_array <<<block_num, thread_num, thread_num * sizeof(double) >>> (shuffled_vec, p0_pD, nx);
					sum_over_array <<<1, block_num, block_num * sizeof(double) >>> (p0_pD, grad_i + i, block_num);
				}
				else {
					sum_over_array <<<block_num, thread_num, thread_num * sizeof(double) >>> (shuffled_vec + nx/2, p0_pD, nx/2);
					sum_over_array <<<1, block_num, block_num * sizeof(double) >>> (p0_pD, grad_i + j, block_num);
				}

				swap_vec = old_vec;
				old_vec = shuffled_vec;
				shuffled_vec = swap_vec;
			}
			else if (i == j) {
				sum_over_array<<<block_num, thread_num, thread_num * sizeof(double)>>>(old_vec, p0_pD, nx);
				sum_over_array<<<1, block_num, block_num * sizeof(double)>>>(p0_pD, grad_i + i, block_num);
			}

			// if the mutation state of the next gene is stored on the current state_copy, make a bit shift to the right
			// else state_copy becomes the next integer stored in the given state (x >> 5  <=> x // 32, x & 31 <=> x % 32)
			if ((j + 1) & 31){
				state_copy >>= 1;
			}
			else {
				state_copy = state->parts[(j + 1) >> 5];
			}
		}
	}
}

__global__ void array_exp(double *arr, int size) {
	int stride = blockDim.x * gridDim.x;
	int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = cuda_index; i < size; i += stride) {
		arr[i] = exp(arr[i]);
	}
}

__global__ void add_to_score(double *score, double *pth_end){
	const int cuda_index = blockIdx.x * blockDim.x + threadIdx.x;

	if(cuda_index == 0){
		score[0] += log(pth_end[0]);
	}
}


/**
 * this function computes the gradient and score for the current MHN for a given data set using CUDA
 *
 * @param[in] ptheta array containing the theta entries
 * @param[in] n number of genes considered by the MHN, also number of columns/rows of theta
 * @param[in] mutation_data array of States, where each state represents a tumor sample
 * @param[in] data_size number of tumor samples in mutation_data
 * @param[out] grad_out array of size n*n in which the gradient will be stored
 *
 * @return this function returns the score of the current MHN as d double value
*/
double DLL_PREFIX cuda_gradient_and_score(double *ptheta, int n, State *mutation_data, int data_size, double *grad_out) {

	// determine the maximum number of mutations present in a single tumor sample
	int max_mutation_num = 0;
	for (int i = 0; i < data_size; i++) {
		if (get_mutation_num(&mutation_data[i]) > max_mutation_num) max_mutation_num = get_mutation_num(&mutation_data[i]);
	}

	const int nx = 1 << max_mutation_num;

	double *cuda_grad_out, *partial_grad;
	double *p0_pD, *pth, *q, *tmp1, *tmp2;
	double *cuda_ptheta;
	double *cuda_score, score;

	// allocate memory on the GPU
	hipMalloc(&cuda_grad_out, n*n * sizeof(double));
	hipMalloc(&partial_grad, n*n * sizeof(double));
	hipMalloc(&p0_pD, nx * sizeof(double));
	hipMalloc(&pth, nx * sizeof(double));
	hipMalloc(&q, nx * sizeof(double));
	hipMalloc(&tmp1, nx * sizeof(double));
	hipMalloc(&tmp2, nx * sizeof(double));
	hipMalloc(&cuda_ptheta, n*n * sizeof(double));

	hipMalloc(&cuda_score, sizeof(double));

	// copy theta to the GPU
	hipMemcpy(cuda_ptheta, ptheta, n*n * sizeof(double), hipMemcpyHostToDevice);
	
	// initialize the gradient on the GPU with zero
	hipMemset(cuda_grad_out, 0, n*n * sizeof(double));

	// for the functions we need theta in its exponential form
	array_exp<<<32, 64>>>(cuda_ptheta, n*n);

	// compute the gradient for each tumor sample and add them together
	for (int i = 0; i < data_size; i++) {
		cuda_restricted_gradient(cuda_ptheta, &mutation_data[i], n, partial_grad, p0_pD, pth, q, tmp1, tmp2);
		add_arrays<<<32, 64>>>(partial_grad, cuda_grad_out, n*n);

		int mutation_num = get_mutation_num(&mutation_data[i]);
		add_to_score<<<1, 1>>>(cuda_score, &pth[(1 << mutation_num) - 1]);

		hipDeviceSynchronize();
	}

	// copy the results to the CPU
	hipMemcpy(grad_out, cuda_grad_out, n*n * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&score, cuda_score, sizeof(double), hipMemcpyDeviceToHost);

	// free all memory on the GPU
	hipFree(partial_grad);
	hipFree(p0_pD);
	hipFree(pth);
	hipFree(q);
	hipFree(tmp1);
	hipFree(tmp2);
	hipFree(cuda_ptheta);

	hipFree(cuda_score);
	hipFree(cuda_grad_out);

	return score;
}
